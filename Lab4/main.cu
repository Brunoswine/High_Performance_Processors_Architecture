#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <windows.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>

__device__ __constant__ int ORDER[4];
__device__ __constant__ int WIDTH;

__global__ void transform(char* data, char* result) {

	result[blockIdx.y * WIDTH + ORDER[threadIdx.x % 4] * WIDTH / 4 + blockDim.x * blockIdx.x / 4 + threadIdx.x / 4] = data[blockIdx.y * WIDTH + blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void sharedTransform(char* data, char* result) {

	const int order[] = { 3, 1, 0, 2 };
	__shared__ char* memory;

	if (!threadIdx.x) {
		memory = (char*)malloc(WIDTH);
	}

	__syncthreads();

	for (int i = 0; i < WIDTH / blockDim.x; i++) {

		memory[order[threadIdx.x % 4] * WIDTH / 4 + blockDim.x * i / 4 + threadIdx.x / 4] = data[blockIdx.x * WIDTH + i * blockDim.x + threadIdx.x];
	}

	__syncthreads();


	for (int i = 0; i < WIDTH / blockDim.x; i++) {

		result[blockIdx.x * WIDTH + i * blockDim.x + threadIdx.x] = memory[i * blockDim.x + threadIdx.x];
	}

	__syncthreads();

	if (!threadIdx.x) {

		free(memory);
	}

}

__global__ void testTransform(int* data, int* result) {

	__shared__ char memory[4][128];
	char buffer[4];

	*(int*)buffer = data[blockIdx.y * WIDTH + blockIdx.x * WIDTH / 512 + threadIdx.y * 128 + threadIdx.x];

	memory[0][32 * threadIdx.y + threadIdx.x] = buffer[2];
	memory[1][32 * threadIdx.y + threadIdx.x] = buffer[1];
	memory[2][32 * threadIdx.y + threadIdx.x] = buffer[3];
	memory[3][32 * threadIdx.y + threadIdx.x] = buffer[0];

	__syncthreads();

	result[blockIdx.y * WIDTH + blockIdx.x * WIDTH / 512 + threadIdx.y * 128 + threadIdx.x] = *(int*)memory[threadIdx.y][4*threadIdx.x];
}

inline
hipError_t CUDA_CALL(hipError_t result)
{
	if (result != hipSuccess)
		std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
	return result;
}
inline
hiprandStatus_t CURAND_CALL(hiprandStatus_t result)
{
	if (result != HIPRAND_STATUS_SUCCESS)
		std::cerr << "CUDA Runtime Error: " << std::endl;
	return result;
}

using namespace std;

class Matrix {

private:
	char* data;
	int height;
	int width;
	int fullSize;

public:

	class sizeEx {};

	Matrix(int height, int width) : height(height), width(width), fullSize(height* width) {

		this->data = new char[fullSize];
		ZeroMemory(this->data, fullSize);

	}

	Matrix(const Matrix& obj) : height(obj.height), width(obj.width), fullSize(obj.fullSize) {

		this->data = new char[fullSize];
		memcpy(this->data, obj.data, fullSize);
	}

	~Matrix() {

		delete[] this->data;
	}

	void fill() {
		for (int i = 0; i < fullSize; i++)
			this->data[i] = '0' + rand() % 10;
	}

	void cudaFill() {

		hiprandGenerator_t gen;
		char* devData;

		CUDA_CALL(hipMalloc(&devData, this->fullSize));

		CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
		CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

		CURAND_CALL(hiprandGenerate(gen, (unsigned int*)devData, this->fullSize / sizeof(unsigned int)));

		CUDA_CALL(hipMemcpy(this->data, devData, this->fullSize, hipMemcpyDeviceToHost));
		CURAND_CALL(hiprandDestroyGenerator(gen));

		CUDA_CALL(hipFree(devData));
	}

	friend std::ostream& operator<<(std::ostream& outStream, const Matrix& obj) {

		for (int i = 0; i < obj.height; i++) {
			for (int j = 0; j < obj.width; j++)
				outStream << obj.data[j + i * obj.width] << " ";
			outStream << endl;
		}

		return outStream;
	}

	bool operator == (const Matrix& obj) {

		if (this->fullSize != obj.fullSize)
			return FALSE;

		for (int i = 0; i < this->height; i++)
			if (memcmp(this->data, obj.data, fullSize))
				return FALSE;

		return TRUE;
	}

	Matrix cpuTransform() const {

		if (this->width % 4)
			throw sizeEx();

		Matrix result(this->height * 4, this->width / 4);
		const int order[] = { 2, 1, 3, 0 };

		DWORD64 startTime = GetTickCount64();

		for (int h = 0; h < this->height; h++) {

			for (int i = 0; i < 4; i++) {

				int tmp = order[i];
				for (int j = 0; j < result.width; j++) {
					result.data[(i + h * 4) * result.width + j] = this->data[h * this->width + tmp + j * 4];
				}
			}

		}

		cout << "CPU  transform elapsed time: " << GetTickCount64() - startTime << " ms" << endl;

		return result;
	}

	Matrix cudaTransform() const {

		if (this->width % 128)
			throw sizeEx();

		Matrix result(this->height * 4, this->width / 4);

		char* dev_data;
		char* dev_result;

		CUDA_CALL(hipMalloc(&dev_data, this->fullSize));
		CUDA_CALL(hipMalloc(&dev_result, result.fullSize));

		dim3 threadsPerBlock = dim3(128);
		dim3 blocksPerGrid = dim3(this->width / 128, this->height);

		hipEvent_t start, stop;
		CUDA_CALL(hipEventCreate(&start));
		CUDA_CALL(hipEventCreate(&stop));

		CUDA_CALL(hipEventRecord(start));

		//data from host to device
		CUDA_CALL(hipMemcpy(dev_data, this->data, this->fullSize, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &this->width, sizeof(int)));
		int order[] = { 3, 1, 0, 2 };
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ORDER), order, sizeof(int) * 4));


		transform << < blocksPerGrid, threadsPerBlock >> > (dev_data, dev_result);

		//result from device to host
		CUDA_CALL(hipMemcpy(result.data, dev_result, this->fullSize, hipMemcpyDeviceToHost));

		CUDA_CALL(hipEventRecord(stop));
		CUDA_CALL(hipEventSynchronize(stop));

		float elapsedTime;
		CUDA_CALL(hipEventElapsedTime(&elapsedTime, start, stop));

		cout << "Cuda transform2 elapsed time: " << (int)elapsedTime << " ms" << endl;

		CUDA_CALL(hipEventDestroy(start));
		CUDA_CALL(hipEventDestroy(stop));

		CUDA_CALL(hipFree(dev_data));
		CUDA_CALL(hipFree(dev_result));

		return result;
	}

	Matrix cudaSharedTransform() const {

		if (this->width % 128)
			throw sizeEx();

		Matrix result(this->height * 4, this->width / 4);

		char* dev_data;
		char* dev_result;

		CUDA_CALL(hipMalloc(&dev_data, this->fullSize));
		CUDA_CALL(hipMalloc(&dev_result, result.fullSize));

		dim3 threadsPerBlock = dim3(128);
		dim3 blocksPerGrid = dim3(this->height);

		hipEvent_t start, stop;
		CUDA_CALL(hipEventCreate(&start));
		CUDA_CALL(hipEventCreate(&stop));

		CUDA_CALL(hipEventRecord(start));

		//data from host to device
		CUDA_CALL(hipMemcpy(dev_data, this->data, this->fullSize, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &this->width, sizeof(int)));

		sharedTransform << < blocksPerGrid, threadsPerBlock >> > (dev_data, dev_result);

		//result from device to host
		CUDA_CALL(hipMemcpy(result.data, dev_result, this->fullSize, hipMemcpyDeviceToHost));

		CUDA_CALL(hipEventRecord(stop));
		CUDA_CALL(hipEventSynchronize(stop));

		float elapsedTime;
		CUDA_CALL(hipEventElapsedTime(&elapsedTime, start, stop));

		cout << "Cuda STransform elapsed time: " << (int)elapsedTime << " ms" << endl;

		CUDA_CALL(hipEventDestroy(start));
		CUDA_CALL(hipEventDestroy(stop));

		CUDA_CALL(hipFree(dev_data));
		CUDA_CALL(hipFree(dev_result));

		return result;
	}

	Matrix cudaTestTransform() const {

		if (this->width % 128)
			throw sizeEx();

		Matrix result(this->height * 4, this->width / 4);

		int* dev_data;
		int* dev_result;

		CUDA_CALL(hipMalloc(&dev_data, this->fullSize));
		CUDA_CALL(hipMalloc(&dev_result, result.fullSize));

		dim3 threadsPerBlock = dim3(32, 4);
		dim3 blocksPerGrid = dim3(this->width / 512, this->height);

		hipEvent_t start, stop;
		CUDA_CALL(hipEventCreate(&start));
		CUDA_CALL(hipEventCreate(&stop));

		CUDA_CALL(hipEventRecord(start));

		//data from host to device
		CUDA_CALL(hipMemcpy(dev_data, this->data, this->fullSize, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &this->width, sizeof(int)));

		testTransform << < blocksPerGrid, threadsPerBlock >> > (dev_data, dev_result);

		//result from device to host
		CUDA_CALL(hipMemcpy(result.data, dev_result, this->fullSize, hipMemcpyDeviceToHost));

		CUDA_CALL(hipEventRecord(stop));
		CUDA_CALL(hipEventSynchronize(stop));

		float elapsedTime;
		CUDA_CALL(hipEventElapsedTime(&elapsedTime, start, stop));

		cout << "Cuda STransform elapsed time: " << (int)elapsedTime << " ms" << endl;

		CUDA_CALL(hipEventDestroy(start));
		CUDA_CALL(hipEventDestroy(stop));

		CUDA_CALL(hipFree(dev_data));
		CUDA_CALL(hipFree(dev_result));

		return result;
	}

	void printSubmatrix(int x0, int y0, int x1, int y1) const {

		if (x0 > x1 || y0 > y1)
			return;

		if (x1 - x0 > this->width || y1 - y0 > this->height)
			throw sizeEx();

		for (int i = y0 - 1; i < y1; i++) {
			for (int j = x0 - 1; j < x1; j++)
				cout << this->data[j + i * this->width] << " ";
			cout << endl;
		}
	}

};

#define HEIGHT 50000
#define WIDTH_AMP 1000

int main() {

	Matrix a(HEIGHT, WIDTH_AMP * 128);

	a.cudaFill();

	try {
		Matrix b = a.cpuTransform();
		Matrix c = a.cudaTransform();
		Matrix d = a.cudaSharedTransform();
		Matrix e = a.cudaTestTransform();


		if (e == b && b == c && b == d)
			cout << "vse ok";
		else cout << "ne vse ok";
	}
	catch (Matrix::sizeEx) {

		cout << "Incorrect matrix size";
	}

	return 0;
}